// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --amap --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "hipblaslt.h"
#include "hipblaslt.h"
// CHECK-NOT: #include "hipblaslt.h"

#if defined(_WIN32) && CUDA_VERSION < 9000
  typedef signed   __int64 int64_t;
  typedef unsigned __int64 uint64_t;
#endif

int main() {
  printf("20. cuBLASLt API to hipBLASLt API synthetic test\n");

  // CHECK: hipblasLtHandle_t blasLtHandle;
  hipblasLtHandle_t blasLtHandle;

  // CHECK: hipblasStatus_t status;
  hipblasStatus_t status;

  // CHECK: hipStream_t stream;
  hipStream_t stream;

  void *A = nullptr;
  void *B = nullptr;
  void *C = nullptr;
  void *D = nullptr;
  void *alpha = nullptr;
  void *beta = nullptr;
  void *workspace = nullptr;
  void *buf = nullptr;
  const char *const_ch = nullptr;

  size_t workspaceSizeInBytes = 0;
  size_t sizeWritten = 0;
  uint64_t rows = 0;
  uint64_t cols = 0;
  int64_t ld = 0;

#if CUDA_VERSION >= 8000
  // CHECK: hipDataType dataType, dataTypeA, dataTypeB, computeType;
  hipDataType dataType, dataTypeA, dataTypeB, computeType;
#endif

#if CUDA_VERSION >= 10010
  // CHECK: hipblasLtMatmulAlgo_t blasLtMatmulAlgo;
  hipblasLtMatmulAlgo_t blasLtMatmulAlgo;

  // CHECK: hipblasLtMatmulDesc_t blasLtMatmulDesc;
  hipblasLtMatmulDesc_t blasLtMatmulDesc;

  // CHECK: hipblasLtMatrixTransformDesc_t blasLtMatrixTransformDesc;
  hipblasLtMatrixTransformDesc_t blasLtMatrixTransformDesc;

  // CHECK: hipblasLtMatmulPreference_t blasLtMatmulPreference;
  hipblasLtMatmulPreference_t blasLtMatmulPreference;

  // CHECK: hipblasLtMatrixLayout_t blasLtMatrixLayout, Adesc, Bdesc, Cdesc, Ddesc;
  hipblasLtMatrixLayout_t blasLtMatrixLayout, Adesc, Bdesc, Cdesc, Ddesc;

  // CHECK: hipblasLtOrder_t blasLtOrder;
  // CHECK-NEXT: hipblasLtOrder_t BLASLT_ORDER_COL = HIPBLASLT_ORDER_COL;
  // CHECK-NEXT: hipblasLtOrder_t BLASLT_ORDER_ROW = HIPBLASLT_ORDER_ROW;
  hipblasLtOrder_t blasLtOrder;
  hipblasLtOrder_t BLASLT_ORDER_COL = HIPBLASLT_ORDER_COL;
  hipblasLtOrder_t BLASLT_ORDER_ROW = HIPBLASLT_ORDER_ROW;

  // CHECK: hipblasLtMatrixLayoutAttribute_t blasLtMatrixLayoutAttribute;
  // CHECK-NEXT: hipblasLtMatrixLayoutAttribute_t BLASLT_MATRIX_LAYOUT_TYPE = HIPBLASLT_MATRIX_LAYOUT_TYPE;
  // CHECK-NEXT: hipblasLtMatrixLayoutAttribute_t BLASLT_MATRIX_LAYOUT_ORDER = HIPBLASLT_MATRIX_LAYOUT_ORDER;
  // CHECK-NEXT: hipblasLtMatrixLayoutAttribute_t BLASLT_MATRIX_LAYOUT_ROWS = HIPBLASLT_MATRIX_LAYOUT_ROWS;
  // CHECK-NEXT: hipblasLtMatrixLayoutAttribute_t BLASLT_MATRIX_LAYOUT_COLS = HIPBLASLT_MATRIX_LAYOUT_COLS;
  // CHECK-NEXT: hipblasLtMatrixLayoutAttribute_t BLASLT_MATRIX_LAYOUT_LD = HIPBLASLT_MATRIX_LAYOUT_LD;
  // CHECK-NEXT: hipblasLtMatrixLayoutAttribute_t BLASLT_MATRIX_LAYOUT_BATCH_COUNT = HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT;
  // CHECK-NEXT: hipblasLtMatrixLayoutAttribute_t BLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET = HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET;
  hipblasLtMatrixLayoutAttribute_t blasLtMatrixLayoutAttribute;
  hipblasLtMatrixLayoutAttribute_t BLASLT_MATRIX_LAYOUT_TYPE = HIPBLASLT_MATRIX_LAYOUT_TYPE;
  hipblasLtMatrixLayoutAttribute_t BLASLT_MATRIX_LAYOUT_ORDER = HIPBLASLT_MATRIX_LAYOUT_ORDER;
  hipblasLtMatrixLayoutAttribute_t BLASLT_MATRIX_LAYOUT_ROWS = HIPBLASLT_MATRIX_LAYOUT_ROWS;
  hipblasLtMatrixLayoutAttribute_t BLASLT_MATRIX_LAYOUT_COLS = HIPBLASLT_MATRIX_LAYOUT_COLS;
  hipblasLtMatrixLayoutAttribute_t BLASLT_MATRIX_LAYOUT_LD = HIPBLASLT_MATRIX_LAYOUT_LD;
  hipblasLtMatrixLayoutAttribute_t BLASLT_MATRIX_LAYOUT_BATCH_COUNT = HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT;
  hipblasLtMatrixLayoutAttribute_t BLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET = HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET;

  // CUDA: cublasStatus_t CUBLASWINAPI cublasLtCreate(cublasLtHandle_t* lightHandle);
  // HIP: HIPBLASLT_EXPORT hipblasStatus_t hipblasLtCreate(hipblasLtHandle_t* handle);
  // CHECK: status = hipblasLtCreate(&blasLtHandle);
  status = hipblasLtCreate(&blasLtHandle);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasLtDestroy(cublasLtHandle_t lightHandle);
  // HIP: HIPBLASLT_EXPORT hipblasStatus_t hipblasLtDestroy(const hipblasLtHandle_t handle);
  // CHECK: status = hipblasLtDestroy(blasLtHandle);
  status = hipblasLtDestroy(blasLtHandle);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasLtMatmul(cublasLtHandle_t lightHandle, cublasLtMatmulDesc_t computeDesc, const void* alpha, const void* A, cublasLtMatrixLayout_t Adesc, const void* B, cublasLtMatrixLayout_t Bdesc, const void* beta, const void* C, cublasLtMatrixLayout_t Cdesc, void* D, cublasLtMatrixLayout_t Ddesc, const cublasLtMatmulAlgo_t* algo, void* workspace, size_t workspaceSizeInBytes, cudaStream_t stream);
  // HIP: HIPBLASLT_EXPORT hipblasStatus_t hipblasLtMatmul(hipblasLtHandle_t handle, hipblasLtMatmulDesc_t matmulDesc, const void* alpha, const void* A, hipblasLtMatrixLayout_t Adesc, const void* B, hipblasLtMatrixLayout_t Bdesc, const void* beta, const void* C, hipblasLtMatrixLayout_t Cdesc, void* D, hipblasLtMatrixLayout_t Ddesc, const hipblasLtMatmulAlgo_t* algo, void* workspace, size_t workspaceSizeInBytes, hipStream_t stream);
  // CHECK: status = hipblasLtMatmul(blasLtHandle, blasLtMatmulDesc, alpha, A, Adesc, B, Bdesc, beta, C, Cdesc, D, Ddesc, &blasLtMatmulAlgo, workspace, workspaceSizeInBytes, stream);
  status = hipblasLtMatmul(blasLtHandle, blasLtMatmulDesc, alpha, A, Adesc, B, Bdesc, beta, C, Cdesc, D, Ddesc, &blasLtMatmulAlgo, workspace, workspaceSizeInBytes, stream);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasLtMatrixTransform(cublasLtHandle_t lightHandle, cublasLtMatrixTransformDesc_t transformDesc, const void* alpha, const void* A, cublasLtMatrixLayout_t Adesc, const void* beta, const void* B, cublasLtMatrixLayout_t Bdesc, void* C, cublasLtMatrixLayout_t Cdesc, cudaStream_t stream);
  // HIP: HIPBLASLT_EXPORT hipblasStatus_t hipblasLtMatrixTransform(hipblasLtHandle_t lightHandle, hipblasLtMatrixTransformDesc_t transformDesc, const void* alpha, const void* A, hipblasLtMatrixLayout_t Adesc, const void* beta, const void* B, hipblasLtMatrixLayout_t Bdesc, void* C, hipblasLtMatrixLayout_t Cdesc, hipStream_t stream);
  // CHECK: status = hipblasLtMatrixTransform(blasLtHandle, blasLtMatrixTransformDesc, alpha, A, Adesc, beta, B, Bdesc, C, Cdesc, stream);
  status = hipblasLtMatrixTransform(blasLtHandle, blasLtMatrixTransformDesc, alpha, A, Adesc, beta, B, Bdesc, C, Cdesc, stream);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasLtMatrixLayoutCreate(cublasLtMatrixLayout_t* matLayout, cudaDataType type, uint64_t rows, uint64_t cols, int64_t ld);
  // HIP: HIPBLASLT_EXPORT hipblasStatus_t hipblasLtMatrixLayoutCreate(hipblasLtMatrixLayout_t* matLayout, hipDataType type, uint64_t rows, uint64_t cols, int64_t ld);
  // CHECK: status = hipblasLtMatrixLayoutCreate(&blasLtMatrixLayout, dataType, rows, cols, ld);
  status = hipblasLtMatrixLayoutCreate(&blasLtMatrixLayout, dataType, rows, cols, ld);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasLtMatrixLayoutDestroy(cublasLtMatrixLayout_t matLayout);
  // HIP: HIPBLASLT_EXPORT hipblasStatus_t hipblasLtMatrixLayoutDestroy(const hipblasLtMatrixLayout_t matLayout);
  // CHECK: status = hipblasLtMatrixLayoutDestroy(blasLtMatrixLayout);
  status = hipblasLtMatrixLayoutDestroy(blasLtMatrixLayout);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasLtMatrixLayoutSetAttribute(cublasLtMatrixLayout_t matLayout, cublasLtMatrixLayoutAttribute_t attr, const void* buf, size_t sizeInBytes);
  // HIP: HIPBLASLT_EXPORT hipblasStatus_t hipblasLtMatrixLayoutSetAttribute(hipblasLtMatrixLayout_t matLayout, hipblasLtMatrixLayoutAttribute_t attr, const void* buf, size_t sizeInBytes);
  // CHECK: status = hipblasLtMatrixLayoutSetAttribute(blasLtMatrixLayout, blasLtMatrixLayoutAttribute, buf, workspaceSizeInBytes);
  status = hipblasLtMatrixLayoutSetAttribute(blasLtMatrixLayout, blasLtMatrixLayoutAttribute, buf, workspaceSizeInBytes);

  // CUDA: cublasStatus_t CUBLASWINAPI cublasLtMatrixLayoutGetAttribute(cublasLtMatrixLayout_t matLayout, cublasLtMatrixLayoutAttribute_t attr, void* buf, size_t sizeInBytes, size_t* sizeWritten);
  // HIP: HIPBLASLT_EXPORT hipblasStatus_t hipblasLtMatrixLayoutGetAttribute(hipblasLtMatrixLayout_t matLayout, hipblasLtMatrixLayoutAttribute_t attr, void* buf, size_t sizeInBytes, size_t* sizeWritten);
  // CHECK: status = hipblasLtMatrixLayoutGetAttribute(blasLtMatrixLayout, blasLtMatrixLayoutAttribute, buf, workspaceSizeInBytes, &sizeWritten);
  status = hipblasLtMatrixLayoutGetAttribute(blasLtMatrixLayout, blasLtMatrixLayoutAttribute, buf, workspaceSizeInBytes, &sizeWritten);
#endif

#if CUBLAS_VERSION >= 10200
  // CHECK: hipblasLtPointerMode_t blasLtPointerMode;
  // CHECK-NEXT: hipblasLtPointerMode_t BLASLT_POINTER_MODE_HOST = HIPBLASLT_POINTER_MODE_HOST;
  // CHECK-NEXT: hipblasLtPointerMode_t BLASLT_POINTER_MODE_DEVICE = HIPBLASLT_POINTER_MODE_DEVICE;
  hipblasLtPointerMode_t blasLtPointerMode;
  hipblasLtPointerMode_t BLASLT_POINTER_MODE_HOST = HIPBLASLT_POINTER_MODE_HOST;
  hipblasLtPointerMode_t BLASLT_POINTER_MODE_DEVICE = HIPBLASLT_POINTER_MODE_DEVICE;
#endif

#if CUDA_VERSION >= 11000 && CUBLAS_VERSION >= 11000
  // CHECK: hipblasLtMatrixLayoutOpaque_t blasLtMatrixLayoutOpaque;
  hipblasLtMatrixLayoutOpaque_t blasLtMatrixLayoutOpaque;

  // CHECK: hipblasLtMatmulDescOpaque_t blasLtMatmulDescOpaque;
  hipblasLtMatmulDescOpaque_t blasLtMatmulDescOpaque;

  // CHECK: hipblasLtMatrixTransformDescOpaque_t blasLtMatrixTransformDescOpaque;
  hipblasLtMatrixTransformDescOpaque_t blasLtMatrixTransformDescOpaque;

  // CHECK: hipblasLtMatmulPreferenceOpaque_t blasLtMatmulPreferenceOpaque;
  hipblasLtMatmulPreferenceOpaque_t blasLtMatmulPreferenceOpaque;
#endif

#if CUDA_VERSION >= 11040 && CUBLAS_VERSION >= 11601
  // CHECK: hipblasLtPointerMode_t BLASLT_POINTER_MODE_ALPHA_DEVICE_VECTOR_BETA_HOST = HIPBLASLT_POINTER_MODE_ALPHA_DEVICE_VECTOR_BETA_HOST;
  hipblasLtPointerMode_t BLASLT_POINTER_MODE_ALPHA_DEVICE_VECTOR_BETA_HOST = HIPBLASLT_POINTER_MODE_ALPHA_DEVICE_VECTOR_BETA_HOST;
#endif
  return 0;
}
